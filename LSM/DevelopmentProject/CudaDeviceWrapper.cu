#include "hip/hip_runtime.h"
#include "CudaDevice.h"   
#include "hip/hip_runtime.h"  

template<typename T, typename P> CudaDevice<T, P>::CudaDevice(){
}
  

template<typename T, typename P> bool CudaDevice<T, P>::isCudaAvailable(){

	int numDevice;
	hipGetDeviceCount(&numDevice);

	return (numDevice > 0);
}

template<typename T, typename P> __device__  void DBsearch(Request<T, P>* arrA, int startPos, int endPos, T &key, int &idx){

	if (startPos == endPos){
		idx = startPos;
	}
	else{
		int mid = (endPos + startPos) / 2;

		T Akey = arrA[mid].getKey();

		if (Akey > key){
			DBsearch<T, P>(arrA, startPos, mid - 1, key, idx);
		}
		else if (Akey < key){
			DBsearch<T, P>(arrA, mid + 1, endPos, key, idx);
		}
		else{
			idx = mid;
		}
	}
	
}

 
template<typename T, typename P> __global__ void merge(Request<T, P>* arrA, Request<T, P>* arrB, int* arrIdx, int arrSize){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < arrSize){

		int Apos;

		T key = arrB[tid].getKey();
			 
		DBsearch<T, P>(arrA, 0, arrSize, key, Apos);

		arrIdx[tid] += Apos;
	}
}

template<typename T, typename P> void CudaDevice<T, P>::mergeKernel(Request<T, P>* arrayA, int arrASize, Request<T, P>* arrayB, int* indices, int size){
	
	int *idx_d;
	Request<T, P> *d_arrayA = 0;
	Request<T, P> *d_arrayB = 0;
	int sizeOfArrA = arrASize * sizeof(Request<T, P>);
	int sizeOfArrB = size * sizeof(Request<T, P>);
	
	//allocate memory
	hipMalloc(&d_arrayA, sizeOfArrA);
	hipMalloc(&d_arrayB, sizeOfArrB);
	hipMalloc(&idx_d, (size * sizeof(int)));
	
	
	//copy data from host to device
	hipMemcpy(d_arrayA, arrayA, sizeOfArrA, hipMemcpyHostToDevice);
	hipMemcpy(d_arrayB, arrayB, sizeOfArrB, hipMemcpyHostToDevice);
	hipMemcpy(idx_d, indices, size * sizeof(int), hipMemcpyHostToDevice);

	//invoke kernel 
	int threadPerBlock = 64;
	int numBlocks = (arrASize / threadPerBlock) + 1;

	merge<T, P> << <numBlocks, threadPerBlock >> >(d_arrayA, d_arrayB, idx_d, arrASize);

	//move data back to host
	hipMemcpy(indices, idx_d, (size * sizeof(int)), hipMemcpyDeviceToHost);
	
	//free device memory
	hipFree(d_arrayA);
	hipFree(d_arrayB);
	hipFree(idx_d);
	
}
