#include "hip/hip_runtime.h"
#include "CudaDevice.h"   
#include "hip/hip_runtime.h"  

template<typename T, typename P> CudaDevice<T, P>::CudaDevice(){
}
  

template<typename T, typename P> bool CudaDevice<T, P>::isCudaAvailable(){

	int numDevice;
	hipGetDeviceCount(&numDevice);

	return (numDevice > 0);
}

/*
* Recursion of binary device search
*/
template<typename T, typename P> __device__  void DBsearch(Request<T, P> *arrA, int startPos, int endPos, Request<T, P> &value, int &idx){

	if (startPos == endPos){
		idx = startPos;
	}
	else{
		int mid = (endPos + startPos) / 2;

		T Akey = arrA[mid].getKey();

		if (Akey > value.getKey()){
			DBsearch<T, P>(arrA, startPos, mid - 1, value, idx);
		}
		else if (Akey < value.getKey()){
			DBsearch<T, P>(arrA, mid + 1, endPos, value, idx);
		}
		else{
			idx = mid;
		}
	}
	
}

/*
* Merge B into A by compute where B's position in respect to A
*
* arrSize is always - 1024 * n where n is greater than 1
*/

template<typename T, typename P> __global__ void merge(Request<T, P> *arrA, Request<T, P> *arrB, int *arrIdx, int arrSize){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < arrSize){

		int work = arrSize / 1024;

		int currWorkIdx = tid * work;

		//using binary search to find the index of A
		int Apos = 0;
		DBsearch<T, P>(arrA, 0, arrSize, arrB[currWorkIdx], Apos);

		arrIdx[currWorkIdx] = arrIdx[currWorkIdx] + Apos;
		currWorkIdx = currWorkIdx + 1;
		work = work - 1;

		//process thru all the works
		while (work > 0){
			DBsearch<T, P>(arrA, Apos, arrSize, arrB[currWorkIdx], Apos);

			arrIdx[currWorkIdx] = arrIdx[currWorkIdx] + Apos;
			currWorkIdx = currWorkIdx + 1;

			work = work - 1;
		}
	}
}

template<typename T, typename P> void CudaDevice<T, P>::mergeKernel(Request<T, P> *arrayA, Request<T, P> *arrayB, int *indices, int size){

	int *idx_d;
	Request<T, P> *d_arrayA;
	Request<T, P> *d_arrayB;
	int sizeOfArr = size * sizeof(Request<T, P>);
	 
	//allocate memory
	hipMalloc(&d_arrayA, sizeOfArr);
	hipMalloc(&d_arrayB, sizeOfArr);
	hipMalloc(&idx_d, (size * sizeof(int)));

	//copy data from host to device
	hipMemcpy(d_arrayA, arrayA, sizeOfArr, hipMemcpyHostToDevice);
	hipMemcpy(d_arrayB, arrayB, sizeOfArr, hipMemcpyHostToDevice);
	hipMemcpy(idx_d, indices, size * sizeof(int), hipMemcpyHostToDevice);

	//invoke kernel
	merge<T, P> <<<32, 32>>> (d_arrayA, d_arrayB, idx_d, size);

	//move data back to host
	hipMemcpy(indices, idx_d, (size * sizeof(int)), hipMemcpyDeviceToHost);

	//free device memory
	hipFree(d_arrayA);
	hipFree(d_arrayB);
	hipFree(idx_d);
}
