#include "hip/hip_runtime.h"
#include "CudaMerger.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

__global__ void CMerge(int* arrA, int* arrB, int* arrIdx, int idxSize){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < idxSize){
		
		arrIdx[tid] = arrIdx[tid] + 20;

	}

}


CudaMerger::CudaMerger(){

}

void CudaMerger::merge(int* &arrA, int lenA, int* &arrB, int lenB){
	
	int *d_idx, *idx, *d_arrA, *d_arrB;
	int sizeAarr = lenA * sizeof(int);
	int sizeBarr = lenB * sizeof(int);

	//initialize idx
	idx = new int[lenB];

	for (int i = 0; i < lenB; i++){
		idx[i] = i;
	}

	//allocate data
	hipMalloc(&d_idx, sizeBarr);
	hipMalloc(&d_arrA, sizeAarr);
	hipMalloc(&d_arrB, sizeBarr);

	//copy data from host to device
	hipMemcpy(d_arrA, arrA, sizeAarr, hipMemcpyHostToDevice);
	hipMemcpy(d_arrB, arrB, sizeBarr, hipMemcpyHostToDevice);
	hipMemcpy(d_idx, idx, sizeBarr, hipMemcpyHostToDevice);

	//invoke kernel
	CMerge<<<32, 32>>>(d_arrA, d_arrB, d_idx, lenB);

	//move data back to host
	hipMemcpy(idx, d_idx, sizeBarr, hipMemcpyDeviceToHost);

	for (int i = 0; i < lenB; i++){
		std::cout << "index: " << idx[i] << std::endl;
	}

	//free resources
	hipFree(d_arrA);
	hipFree(d_arrB);
	hipFree(d_idx);
	delete idx;
}